#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/utils/logger.h"
#include "src/fastertransformer/utils/memory_utils.h"
#include "src/fastertransformer/utils/mpi_utils.h"
#include <hiprand/hiprand_kernel.h>

namespace fastertransformer {

// Modified
void extractSharedPtr(void** ptr, const TensorMetaTransit_t& shared_tensor_metadata) {
    // Extract the pointer
    check_cuda_error(hipIpcOpenMemHandle(ptr, shared_tensor_metadata.recv_tensor_handle, hipIpcMemLazyEnablePeerAccess));
}
void releaseSharedPtr(void* ptr){
    // FT_LOG_INFO("Release ptr : %p", ptr);
    check_cuda_error(hipIpcCloseMemHandle(ptr));
}

// Modified
template<typename T>
int64_t requestTensor(T** ptr, size_t size, int layer_id, int param_idx, int param_type, TcpAgent* tcp_agent, bool try_reuse){
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);

    struct ClientRequest_t req{layer_id, param_type, param_idx, (int)size};
    struct TensorMetaTransit_t rsp;
    if(param_type == 9){
        tcp_agent->tcpSend(&req, sizeof(ClientRequest_t));
        tcp_agent->tcpRecv(&rsp, sizeof(TensorMetaTransit_t));
        return 0;
    }

    // FT_LOG_INFO(mpi::getCommWorldRank(), "requestTensor: %p layer_id=%d, param_idx=%d, param_type=%d",*ptr, layer_id, param_idx, param_type);
    if(try_reuse && param_type >= 6){
        // buffer try reuse
        req.param_type -= 4; // get buffer first
        tcp_agent->tcpSend(&req, sizeof(ClientRequest_t));
        tcp_agent->tcpRecv(&rsp, sizeof(TensorMetaTransit_t));

        if(size * sizeof(T) == rsp.stoage_size){
            // size is same, reuse
            FT_LOG_DEBUG("%p REUSED when layer_id=%d, param_idx=%d, param_type=%d",*ptr, layer_id, param_idx, param_type);
            check_cuda_error(hipMemset(*ptr, 0, rsp.stoage_size));
            return 0;
        }
        // size mistch, PC realloc, FT close memhandle first
        releaseSharedPtr(*ptr);
        req.param_type += 4; // alloc afterwards
    }

    tcp_agent->tcpSend(&req, sizeof(ClientRequest_t));
    tcp_agent->tcpRecv(&rsp, sizeof(TensorMetaTransit_t));

    FT_CHECK_WITH_INFO(size * sizeof(T) == rsp.stoage_size,
        fmtstr("%d==%d mismatch when layer_id=%d, param_idx=%d, param_type=%d", size * sizeof(T), rsp.stoage_size, layer_id, param_idx, param_type));

    int64_t offset = 0;
    extractSharedPtr((void**)ptr, rsp);
    offset = rsp.storage_offset / sizeof(T);
    *ptr = (*ptr) + offset;

    return offset;
}

template int64_t requestTensor(float** ptr, size_t size, int layer_id, int param_idx, int param_type, TcpAgent* tcp_agent, bool try_reuse);
template int64_t requestTensor(half** ptr, size_t size, int layer_id, int param_idx, int param_type, TcpAgent* tcp_agent, bool try_reuse);
#ifdef ENABLE_BF16
template int64_t requestTensor(__hip_bfloat16** ptr, size_t size, int layer_id, int param_idx, int param_type, TcpAgent* tcp_agent, bool try_reuse);
#endif
template int64_t requestTensor(uint16_t** ptr, size_t size, int layer_id, int param_idx, int param_type, TcpAgent* tcp_agent, bool try_reuse);
template int64_t requestTensor(int** ptr, size_t size, int layer_id, int param_idx, int param_type, TcpAgent* tcp_agent, bool try_reuse);
template int64_t requestTensor(bool** ptr, size_t size, int layer_id, int param_idx, int param_type, TcpAgent* tcp_agent, bool try_reuse);
template int64_t requestTensor(char** ptr, size_t size, int layer_id, int param_idx, int param_type, TcpAgent* tcp_agent, bool try_reuse);
template int64_t requestTensor(int8_t** ptr, size_t size, int layer_id, int param_idx, int param_type, TcpAgent* tcp_agent, bool try_reuse);


template<typename T>
void deviceMalloc(T** ptr, size_t size, bool is_random_initialize)
{
    FT_CHECK_WITH_INFO(size >= 0, "Ask deviceMalloc size " + std::to_string(size) + "< 0 is invalid.");
    check_cuda_error(hipMalloc((void**)(ptr), sizeof(T) * size));
    // hipMemcpy((void*)ptr, (void*)ptr, sizeof(T) * size, hipMemcpyDeviceToDevice);
    if (is_random_initialize) {
        cudaRandomUniform(*ptr, size);
    }
}

template void deviceMalloc(float** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(half** ptr, size_t size, bool is_random_initialize);
#ifdef ENABLE_BF16
template void deviceMalloc(__hip_bfloat16** ptr, size_t size, bool is_random_initialize);
#endif
template void deviceMalloc(uint16_t** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(int** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(bool** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(char** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(int8_t** ptr, size_t size, bool is_random_initialize);

template<typename T>
void deviceMemSetZero(T* ptr, int size)
{
    check_cuda_error(hipMemset(static_cast<void*>(ptr), 0, sizeof(T) * size));
}

template void deviceMemSetZero(float* ptr, int size);
template void deviceMemSetZero(half* ptr, int size);
template void deviceMemSetZero(int* ptr, int size);
template void deviceMemSetZero(uint32_t* ptr, int size);
template void deviceMemSetZero(bool* ptr, int size);

template<typename T>
void deviceFree(T*& ptr)
{
    if (ptr != NULL) {
        check_cuda_error(hipFree(ptr));
        ptr = NULL;
    }
}

template void deviceFree(float*& ptr);
template void deviceFree(half*& ptr);
#ifdef ENABLE_BF16
template void deviceFree(__hip_bfloat16*& ptr);
#endif
template void deviceFree(unsigned short*& ptr);
template void deviceFree(int*& ptr);
template void deviceFree(bool*& ptr);
template void deviceFree(char*& ptr);
template void deviceFree(int8_t*& ptr);

template<typename T>
void deviceFill(T* devptr, int size, T value, hipStream_t stream)
{
    T* arr = new T[size];
    std::fill(arr, arr + size, value);
    check_cuda_error(hipMemcpyAsync(devptr, arr, sizeof(T) * size, hipMemcpyHostToDevice, stream));
    delete[] arr;
}

template void deviceFill(float* devptr, int size, float value, hipStream_t stream);
template void deviceFill(half* devptr, int size, half value, hipStream_t stream);
#ifdef ENABLE_BF16
template void deviceFill(__hip_bfloat16* devptr, int size, __hip_bfloat16 value, hipStream_t stream);
#endif
template void deviceFill(int* devptr, int size, int value, hipStream_t stream);
template void deviceFill(bool* devptr, int size, bool value, hipStream_t stream);

template<typename T>
void cudaD2Hcpy(T* tgt, const T* src, const int size)
{
    // FT_LOG_INFO("target=%p, src=%p", tgt, src);
    check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyDeviceToHost));
}

template void cudaD2Hcpy(float* tgt, const float* src, int size);
template void cudaD2Hcpy(half* tgt, const half* src, int size);
#ifdef ENABLE_BF16
template void cudaD2Hcpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, int size);
#endif
template void cudaD2Hcpy(int* tgt, const int* src, int size);
template void cudaD2Hcpy(bool* tgt, const bool* src, int size);
template void cudaD2Hcpy(unsigned long long* tgt, const unsigned long long* src, int size);
template void cudaD2Hcpy(unsigned int* tgt, const unsigned int* src, int size);

template<typename T>
void cudaH2Dcpy(T* tgt, const T* src, const int size)
{
    check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyHostToDevice));
}

template void cudaH2Dcpy(float* tgt, const float* src, int size);
template void cudaH2Dcpy(half* tgt, const half* src, int size);
#ifdef ENABLE_BF16
template void cudaH2Dcpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, int size);
#endif
template void cudaH2Dcpy(int* tgt, const int* src, int size);
template void cudaH2Dcpy(bool* tgt, const bool* src, int size);
template void cudaH2Dcpy(unsigned long long* tgt, const unsigned long long* src, int size);
template void cudaH2Dcpy(unsigned int* tgt, const unsigned int* src, int size);

template<typename T>
void cudaD2Dcpy(T* tgt, const T* src, const int size)
{
    check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyDeviceToDevice));
}

template void cudaD2Dcpy(float* tgt, const float* src, int size);
template void cudaD2Dcpy(half* tgt, const half* src, int size);
#ifdef ENABLE_BF16
template void cudaD2Dcpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, int size);
#endif
template void cudaD2Dcpy(int* tgt, const int* src, int size);
template void cudaD2Dcpy(bool* tgt, const bool* src, int size);
template void cudaD2Dcpy(int8_t* tgt, const int8_t* src, int size);
template void cudaD2Dcpy(unsigned long long* tgt, const unsigned long long* src, int size);

template<typename T>
void cudaAutoCpy(T* tgt, const T* src, const int size, hipStream_t stream)
{
    if (stream != NULL) {
        check_cuda_error(hipMemcpyAsync(tgt, src, sizeof(T) * size, hipMemcpyDefault, stream));
    }
    else {
        check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyDefault));
    }
}

template void cudaAutoCpy(float* tgt, const float* src, int size, hipStream_t stream);
template void cudaAutoCpy(half* tgt, const half* src, int size, hipStream_t stream);
#ifdef ENABLE_BF16
template void cudaAutoCpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, int size, hipStream_t stream);
#endif
template void cudaAutoCpy(int* tgt, const int* src, int size, hipStream_t stream);
template void cudaAutoCpy(bool* tgt, const bool* src, int size, hipStream_t stream);
template void cudaAutoCpy(int8_t* tgt, const int8_t* src, int size, hipStream_t stream);
template void cudaAutoCpy(uint* tgt, const uint* src, int size, hipStream_t stream);
template void cudaAutoCpy(unsigned long long* tgt, const unsigned long long* src, int size, hipStream_t stream);

template void cudaAutoCpy(float const** tgt, float const* const* src, int size, hipStream_t stream);
template void cudaAutoCpy(half const** tgt, half const* const* src, int size, hipStream_t stream);
#ifdef ENABLE_BF16
template void cudaAutoCpy(__hip_bfloat16 const** tgt, __hip_bfloat16 const* const* src, int size, hipStream_t stream);
#endif
template void cudaAutoCpy(int const** tgt, int const* const* src, int size, hipStream_t stream);
template void cudaAutoCpy(bool const** tgt, bool const* const* src, int size, hipStream_t stream);
template void cudaAutoCpy(int8_t const** tgt, int8_t const* const* src, int size, hipStream_t stream);
template void
cudaAutoCpy(unsigned long long const** tgt, unsigned long long const* const* src, int size, hipStream_t stream);

template<typename T>
__global__ void cuda_random_uniform_kernel(T* buffer, const int size)
{
    const int     idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t local_state;
    hiprand_init((unsigned long long int)1337, idx, 0, &local_state);
    for (int index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = (T)(hiprand_uniform(&local_state) * 0.2f - 0.1f);
    }
}

template<typename T>
__global__ void cuda_random_uniform_kernel(int* buffer, const int size)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = 0;
    }
}

template<typename T>
__global__ void cuda_random_uniform_kernel(bool* buffer, const int size)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = false;
    }
}

template<typename T>
__global__ void cuda_random_uniform_kernel(char* buffer, const int size)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = '\0';
    }
}

template<typename T>
void cudaRandomUniform(T* buffer, const int size)
{
    cuda_random_uniform_kernel<T><<<256, 256>>>(buffer, size);
}

template void cudaRandomUniform(float* buffer, const int size);
template void cudaRandomUniform(half* buffer, const int size);
#ifdef ENABLE_BF16
template void cudaRandomUniform(__hip_bfloat16* buffer, const int size);
#endif
template void cudaRandomUniform(int* buffer, const int size);
template void cudaRandomUniform(bool* buffer, const int size);
template void cudaRandomUniform(char* buffer, const int size);

template<typename T_IN, typename T_OUT>
__host__ __device__ inline T_OUT convert_to_type(T_IN val)
{
    return (T_OUT)val;
}

#ifdef ENABLE_BF16
template<>
__host__ __device__ inline __hip_bfloat16 convert_to_type<float, __hip_bfloat16>(float val)
{
    return __float2bfloat16(val);
}

template<>
__host__ __device__ inline __hip_bfloat16 convert_to_type<half, __hip_bfloat16>(half val)
{
    return __float2bfloat16(__half2float(val));
}

template<>
__host__ __device__ inline float convert_to_type<__hip_bfloat16, float>(__hip_bfloat16 val)
{
    return __bfloat162float(val);
}

template<>
__host__ __device__ inline half convert_to_type<__hip_bfloat16, half>(__hip_bfloat16 val)
{
    return __float2half(__bfloat162float(val));
}
#endif  // ENABLE_BF16

template<typename T, typename T_IN>
int loadWeightFromBinFunc(T* ptr, std::vector<size_t> shape, std::string filename)
{
    if (shape.size() > 2) {
        printf("[ERROR] shape should have less than two dims \n");
        return -1;
    }
    size_t dim0 = shape[0], dim1 = 1;
    if (shape.size() == 2) {
        dim1 = shape[1];
    }
    size_t size = dim0 * dim1;
    if (size == 0) {
        FT_LOG_WARNING("shape is zero, skip loading weight from file %s \n", filename.c_str());
        return 0;
    }
    std::vector<T_IN> host_array(size);
    std::ifstream     in(filename, std::ios::in | std::ios::binary);
    if (!in.is_open()) {
        FT_LOG_WARNING("file %s cannot be opened, loading model fails! \n", filename.c_str());
        return 0;
    }

    size_t loaded_data_size = sizeof(T_IN) * size;
    in.seekg(0, in.end);
    in.seekg(0, in.beg);

    FT_LOG_DEBUG("Read " + std::to_string(loaded_data_size) + " bytes from " + filename);
    in.read((char*)host_array.data(), loaded_data_size);

    size_t in_get_size = in.gcount();
    if (in_get_size != loaded_data_size) {
        FT_LOG_WARNING("file %s only has %ld, but request %ld, loading model fails! \n",
                       filename.c_str(),
                       in_get_size,
                       loaded_data_size);
        return 0;
    }

    if (std::is_same<T, T_IN>::value == true) {
        cudaH2Dcpy(ptr, (T*)host_array.data(), size);
    }
    else {
        T_IN* ptr_2 = nullptr;
        deviceMalloc(&ptr_2, size, false);
        cudaH2Dcpy(ptr_2, host_array.data(), size);
        invokeCudaD2DcpyConvert(ptr, ptr_2, size);
        deviceFree(ptr_2);
    }
    in.close();
    return 0;
}

template int loadWeightFromBinFunc<float, float>(float* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<half, float>(half* ptr, std::vector<size_t> shape, std::string filename);
#ifdef ENABLE_BF16
template int
loadWeightFromBinFunc<__hip_bfloat16, float>(__hip_bfloat16* ptr, std::vector<size_t> shape, std::string filename);
#endif
template int loadWeightFromBinFunc<float, half>(float* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<half, half>(half* ptr, std::vector<size_t> shape, std::string filename);
#ifdef ENABLE_BF16
template int
loadWeightFromBinFunc<__hip_bfloat16, half>(__hip_bfloat16* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<float, __hip_bfloat16>(float* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<half, __hip_bfloat16>(half* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<__hip_bfloat16, __hip_bfloat16>(__hip_bfloat16*      ptr,
                                                                 std::vector<size_t> shape,
                                                                 std::string         filename);
#endif  // ENABLE_BF16

template<typename T>
int loadWeightFromBin(T* ptr, std::vector<size_t> shape, std::string filename, FtCudaDataType model_file_type)
{
    switch (model_file_type) {
        case FtCudaDataType::FP32:
            loadWeightFromBinFunc<T, float>(ptr, shape, filename);
            break;
        case FtCudaDataType::FP16:
            loadWeightFromBinFunc<T, half>(ptr, shape, filename);
            break;
#ifdef ENABLE_BF16
        case FtCudaDataType::BF16:
            loadWeightFromBinFunc<T, __hip_bfloat16>(ptr, shape, filename);
            break;
#endif
        default:
            FT_LOG_ERROR("Does not support FtCudaDataType=%d", model_file_type);
            FT_CHECK(false);
    }
    return 0;
}

template int
loadWeightFromBin(float* ptr, std::vector<size_t> shape, std::string filename, FtCudaDataType model_file_type);
template int
loadWeightFromBin(half* ptr, std::vector<size_t> shape, std::string filename, FtCudaDataType model_file_type);
#ifdef ENABLE_BF16
template int
loadWeightFromBin(__hip_bfloat16* ptr, std::vector<size_t> shape, std::string filename, FtCudaDataType model_file_type);
#endif

template<typename T_IN, typename T_OUT>
__global__ void cudaD2DcpyConvert(T_OUT* dst, const T_IN* src, const int size)
{
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x) {
        dst[tid] = convert_to_type<T_IN, T_OUT>(src[tid]);
    }
}

template<typename T_IN, typename T_OUT>
void invokeCudaD2DcpyConvert(T_OUT* tgt, const T_IN* src, const int size, hipStream_t stream)
{
    cudaD2DcpyConvert<<<256, 256, 0, stream>>>(tgt, src, size);
}

template void invokeCudaD2DcpyConvert(float* tgt, const float* src, const int size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(half* tgt, const float* src, const int size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, const half* src, const int size, hipStream_t stream);

#ifdef ENABLE_BF16
template void invokeCudaD2DcpyConvert(__hip_bfloat16* tgt, const float* src, const int size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, const __hip_bfloat16* src, const int size, hipStream_t stream);
#endif  // ENABLE_BF16

void invokeCudaD2DcpyHalf2Float(float* dst, half* src, const int size, hipStream_t stream)
{
    invokeCudaD2DcpyConvert(dst, src, size, stream);
}

void invokeCudaD2DcpyFloat2Half(half* dst, float* src, const int size, hipStream_t stream)
{
    invokeCudaD2DcpyConvert(dst, src, size, stream);
}

template<typename T>
void saveToBinary(const T* ptr, const int size, std::string filename)
{

    std::vector<T> h_ptr(size);
    cudaD2Hcpy(h_ptr.data(), ptr, size);
    std::vector<float> float_ptr(size);
    for (int i = 0; i < size; i++) {
        float_ptr[i] = (float)h_ptr[i];
    }

    std::ofstream out(filename, std::ios::out | std::ios::binary);
    FT_CHECK_WITH_INFO(out.is_open(), "Fail to open file " + filename);

    out.write((char*)float_ptr.data(), size * sizeof(float));
}

template void saveToBinary(const float* ptr, const int size, std::string filename);
template void saveToBinary(const half* ptr, const int size, std::string filename);
#ifdef ENABLE_BF16
template void saveToBinary(const __hip_bfloat16* ptr, const int size, std::string filename);
#endif  // ENABLE_BF16

template<>
void saveToBinary(const int* ptr, const int size, std::string filename)
{
    std::vector<int> h_ptr(size);
    cudaD2Hcpy(h_ptr.data(), ptr, size);
    std::ofstream out(filename, std::ios::out | std::ios::binary);
    FT_CHECK_WITH_INFO(out.is_open(), "Fail to open file " + filename);
    out.write((char*)h_ptr.data(), size * sizeof(int));
}

template<typename T_IN, typename T_fake_type>
__global__ void fakeCast(T_IN* input_ptr, const size_t size)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        T_fake_type tmp_val = (T_fake_type)((float)input_ptr[i]);
        tmp_val             = tmp_val * (T_fake_type)(1.0f);
        input_ptr[i]        = (T_IN)((float)tmp_val);
    }
}

template<typename T_IN, typename T_fake_type>
void invokeFakeCast(T_IN* input_ptr, const size_t size, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((size + 255) / 256);
    fakeCast<T_IN, T_fake_type><<<grid, block, 0, stream>>>(input_ptr, size);
}

#ifdef ENABLE_BF16
template void invokeFakeCast<float, __hip_bfloat16>(float* input_ptr, const size_t size, hipStream_t stream);
template void
invokeFakeCast<__hip_bfloat16, __hip_bfloat16>(__hip_bfloat16* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<half, __hip_bfloat16>(half* input_ptr, const size_t size, hipStream_t stream);
#endif
template void invokeFakeCast<float, half>(float* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<float, float>(float* input_ptr, const size_t size, hipStream_t stream);

}  // namespace fastertransformer
