#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/utils/logger.h"
#include "src/utils/memory_utils.h"
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <sstream>

#ifdef AWS_SDK_ON
#include <aws/core/Aws.h>
#include <aws/s3/S3Client.h>
#include <aws/s3/model/GetObjectRequest.h>
#endif

namespace fastertransformer {

// Modified
void extractSharedPtr(void** ptr, const TensorMetaTransit_t& shared_tensor_metadata) {
    // Extract the pointer
    hipError_t err = hipIpcOpenMemHandle(ptr, shared_tensor_metadata.recv_tensor_handle, hipIpcMemLazyEnablePeerAccess);
    if (err != hipSuccess) {
      perror("extract_shared_cache");
      exit(EXIT_FAILURE);
    }
}

std::string MyGetBucketNameFromS3Uri(const std::string& s3Uri) {
    std::string bucketName;
    std::size_t pos = s3Uri.find("//");
    if (pos != std::string::npos) {
        pos += 2;
        std::size_t endPos = s3Uri.find("/", pos);
        if (endPos != std::string::npos) {
            bucketName = s3Uri.substr(pos, endPos - pos);
        } else {
            bucketName = s3Uri.substr(pos);
        }
    }
    return bucketName;
}

std::string MyGetObjectKeyFromS3Uri(const std::string& s3Uri) {
    std::string objectKey;
    std::size_t pos = s3Uri.find("//");
    if (pos != std::string::npos) {
        pos = s3Uri.find("/", pos + 2);
        if (pos != std::string::npos) {
            objectKey = s3Uri.substr(pos + 1);
        }
    }
    return objectKey;
}


template<typename T>
void deviceMalloc(T** ptr, size_t size, bool is_random_initialize)
{
    FT_CHECK_WITH_INFO(size >= 0, "Ask deviceMalloc size " + std::to_string(size) + "< 0 is invalid.");
    check_cuda_error(hipMalloc((void**)(ptr), sizeof(T) * size));
    if (is_random_initialize) {
        cudaRandomUniform(*ptr, size);
    }
}

template void deviceMalloc(float** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(half** ptr, size_t size, bool is_random_initialize);
#ifdef ENABLE_BF16
template void deviceMalloc(__hip_bfloat16** ptr, size_t size, bool is_random_initialize);
#endif
template void deviceMalloc(uint16_t** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(int** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(bool** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(char** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(int8_t** ptr, size_t size, bool is_random_initialize);

template<typename T>
void deviceMemSetZero(T* ptr, int size)
{
    check_cuda_error(hipMemset(static_cast<void*>(ptr), 0, sizeof(T) * size));
}

template void deviceMemSetZero(float* ptr, int size);
template void deviceMemSetZero(half* ptr, int size);
template void deviceMemSetZero(int* ptr, int size);
template void deviceMemSetZero(uint32_t* ptr, int size);
template void deviceMemSetZero(bool* ptr, int size);

template<typename T>
void deviceFree(T*& ptr)
{
    if (ptr != NULL) {
        check_cuda_error(hipFree(ptr));
        ptr = NULL;
    }
}

template void deviceFree(float*& ptr);
template void deviceFree(half*& ptr);
#ifdef ENABLE_BF16
template void deviceFree(__hip_bfloat16*& ptr);
#endif
template void deviceFree(unsigned short*& ptr);
template void deviceFree(int*& ptr);
template void deviceFree(bool*& ptr);
template void deviceFree(char*& ptr);
template void deviceFree(int8_t*& ptr);

template<typename T>
void deviceFill(T* devptr, int size, T value, hipStream_t stream)
{
    T* arr = new T[size];
    std::fill(arr, arr + size, value);
    check_cuda_error(hipMemcpyAsync(devptr, arr, sizeof(T) * size, hipMemcpyHostToDevice, stream));
    delete[] arr;
}

template void deviceFill(float* devptr, int size, float value, hipStream_t stream);
template void deviceFill(half* devptr, int size, half value, hipStream_t stream);
#ifdef ENABLE_BF16
template void deviceFill(__hip_bfloat16* devptr, int size, __hip_bfloat16 value, hipStream_t stream);
#endif
template void deviceFill(int* devptr, int size, int value, hipStream_t stream);
template void deviceFill(bool* devptr, int size, bool value, hipStream_t stream);

template<typename T>
void cudaD2Hcpy(T* tgt, const T* src, const int size)
{
    check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyDeviceToHost));
}

template void cudaD2Hcpy(float* tgt, const float* src, int size);
template void cudaD2Hcpy(half* tgt, const half* src, int size);
#ifdef ENABLE_BF16
template void cudaD2Hcpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, int size);
#endif
template void cudaD2Hcpy(int* tgt, const int* src, int size);
template void cudaD2Hcpy(bool* tgt, const bool* src, int size);
template void cudaD2Hcpy(unsigned long long* tgt, const unsigned long long* src, int size);
template void cudaD2Hcpy(unsigned int* tgt, const unsigned int* src, int size);

template<typename T>
void cudaH2Dcpy(T* tgt, const T* src, const int size)
{
    check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyHostToDevice));
}

template void cudaH2Dcpy(float* tgt, const float* src, int size);
template void cudaH2Dcpy(half* tgt, const half* src, int size);
#ifdef ENABLE_BF16
template void cudaH2Dcpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, int size);
#endif
template void cudaH2Dcpy(int* tgt, const int* src, int size);
template void cudaH2Dcpy(bool* tgt, const bool* src, int size);
template void cudaH2Dcpy(unsigned long long* tgt, const unsigned long long* src, int size);
template void cudaH2Dcpy(unsigned int* tgt, const unsigned int* src, int size);

template<typename T>
void cudaD2Dcpy(T* tgt, const T* src, const int size)
{
    check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyDeviceToDevice));
}

template void cudaD2Dcpy(float* tgt, const float* src, int size);
template void cudaD2Dcpy(half* tgt, const half* src, int size);
#ifdef ENABLE_BF16
template void cudaD2Dcpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, int size);
#endif
template void cudaD2Dcpy(int* tgt, const int* src, int size);
template void cudaD2Dcpy(bool* tgt, const bool* src, int size);
template void cudaD2Dcpy(int8_t* tgt, const int8_t* src, int size);
template void cudaD2Dcpy(unsigned long long* tgt, const unsigned long long* src, int size);

template<typename T>
void cudaAutoCpy(T* tgt, const T* src, const int size, hipStream_t stream)
{
    if (stream != NULL) {
        check_cuda_error(hipMemcpyAsync(tgt, src, sizeof(T) * size, hipMemcpyDefault, stream));
    }
    else {
        check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyDefault));
    }
}

template void cudaAutoCpy(float* tgt, const float* src, int size, hipStream_t stream);
template void cudaAutoCpy(half* tgt, const half* src, int size, hipStream_t stream);
#ifdef ENABLE_BF16
template void cudaAutoCpy(__hip_bfloat16* tgt, const __hip_bfloat16* src, int size, hipStream_t stream);
#endif
template void cudaAutoCpy(int* tgt, const int* src, int size, hipStream_t stream);
template void cudaAutoCpy(bool* tgt, const bool* src, int size, hipStream_t stream);
template void cudaAutoCpy(int8_t* tgt, const int8_t* src, int size, hipStream_t stream);
template void cudaAutoCpy(uint* tgt, const uint* src, int size, hipStream_t stream);
template void cudaAutoCpy(unsigned long long* tgt, const unsigned long long* src, int size, hipStream_t stream);

template void cudaAutoCpy(float const** tgt, float const* const* src, int size, hipStream_t stream);
template void cudaAutoCpy(half const** tgt, half const* const* src, int size, hipStream_t stream);
#ifdef ENABLE_BF16
template void cudaAutoCpy(__hip_bfloat16 const** tgt, __hip_bfloat16 const* const* src, int size, hipStream_t stream);
#endif
template void cudaAutoCpy(int const** tgt, int const* const* src, int size, hipStream_t stream);
template void cudaAutoCpy(bool const** tgt, bool const* const* src, int size, hipStream_t stream);
template void cudaAutoCpy(int8_t const** tgt, int8_t const* const* src, int size, hipStream_t stream);
template void
cudaAutoCpy(unsigned long long const** tgt, unsigned long long const* const* src, int size, hipStream_t stream);

template<typename T>
__global__ void cuda_random_uniform_kernel(T* buffer, const int size)
{
    const int     idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t local_state;
    hiprand_init((unsigned long long int)1337, idx, 0, &local_state);
    for (int index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = (T)(hiprand_uniform(&local_state) * 0.2f - 0.1f);
    }
}

template<typename T>
__global__ void cuda_random_uniform_kernel(int* buffer, const int size)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = 0;
    }
}

template<typename T>
__global__ void cuda_random_uniform_kernel(bool* buffer, const int size)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = false;
    }
}

template<typename T>
__global__ void cuda_random_uniform_kernel(char* buffer, const int size)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int index = idx; index < size; index += blockDim.x * gridDim.x) {
        buffer[index] = '\0';
    }
}

template<typename T>
void cudaRandomUniform(T* buffer, const int size)
{
    cuda_random_uniform_kernel<T><<<256, 256>>>(buffer, size);
}

template void cudaRandomUniform(float* buffer, const int size);
template void cudaRandomUniform(half* buffer, const int size);
#ifdef ENABLE_BF16
template void cudaRandomUniform(__hip_bfloat16* buffer, const int size);
#endif
template void cudaRandomUniform(int* buffer, const int size);
template void cudaRandomUniform(bool* buffer, const int size);
template void cudaRandomUniform(char* buffer, const int size);

template<typename T_IN, typename T_OUT>
__host__ __device__ inline T_OUT convert_to_type(T_IN val)
{
    return (T_OUT)val;
}

#ifdef ENABLE_BF16
template<>
__host__ __device__ inline __hip_bfloat16 convert_to_type<float, __hip_bfloat16>(float val)
{
    return __float2bfloat16(val);
}

template<>
__host__ __device__ inline __hip_bfloat16 convert_to_type<half, __hip_bfloat16>(half val)
{
    return __float2bfloat16(__half2float(val));
}

template<>
__host__ __device__ inline float convert_to_type<__hip_bfloat16, float>(__hip_bfloat16 val)
{
    return __bfloat162float(val);
}

template<>
__host__ __device__ inline half convert_to_type<__hip_bfloat16, half>(__hip_bfloat16 val)
{
    return __float2half(__bfloat162float(val));
}
#endif  // ENABLE_BF16

template<typename T, typename T_IN>
int loadWeightFromBinFunc(T* ptr, std::vector<size_t> shape, std::string filename)
{
    if (shape.size() > 2) {
        printf("[ERROR] shape should have less than two dims \n");
        return -1;
    }
    size_t dim0 = shape[0], dim1 = 1;
    if (shape.size() == 2) {
        dim1 = shape[1];
    }
    size_t size = dim0 * dim1;
    if (size == 0) {
        FT_LOG_WARNING("shape is zero, skip loading weight from file %s \n", filename.c_str());
        return 0;
    }
    std::vector<T_IN> host_array(size);
    size_t loaded_data_size = sizeof(T_IN) * size;
    size_t in_get_size = 0;

#ifdef AWS_SDK_ON
    Aws::SDKOptions options;
    if (filename.find("s3://") == 0) {
        std::string s3BucketName = MyGetBucketNameFromS3Uri(filename);
        std::string s3ObjectName = MyGetObjectKeyFromS3Uri(filename);
        Aws::InitAPI(options);

        size_t s3ObjectSize = 0;
        Aws::S3::S3Client s3Client;
        Aws::S3::Model::GetObjectRequest objectRequest;
        objectRequest.SetBucket(s3BucketName.c_str());
        objectRequest.SetKey(s3ObjectName.c_str());
        auto s3GetObjectOutcome = s3Client.GetObject(objectRequest);
        if (s3GetObjectOutcome.IsSuccess()) {
            auto& s3ObjectStream = s3GetObjectOutcome.GetResultWithOwnership().GetBody();
            s3ObjectSize = s3GetObjectOutcome.GetResultWithOwnership().GetContentLength();

            s3ObjectStream.seekg(0, s3ObjectStream.end);
            s3ObjectStream.seekg(0, s3ObjectStream.beg);

            s3ObjectStream.read((char*)host_array.data(), s3ObjectSize);
            in_get_size = s3ObjectStream.gcount();
        } else {
            FT_LOG_WARNING("S3 object %s cannot be downloaded, loading model fails! \n", s3ObjectName.c_str());
            return 0;
        }
    } else {
#endif
        std::ifstream     in = std::ifstream(filename, std::ios::in | std::ios::binary);

        if (!in.is_open()) {
            FT_LOG_WARNING("file %s cannot be opened, loading model fails! \n", filename.c_str());
            return 0;
        }

        in.seekg(0, in.end);
        in.seekg(0, in.beg);

        // FT_LOG_DEBUG("Read " + std::to_string(loaded_data_size) + " bytes from " + filename);
        in.read((char*)host_array.data(), loaded_data_size);
        in_get_size = in.gcount();
        in.close();
#ifdef AWS_SDK_ON
    }
#endif

    if (in_get_size != loaded_data_size) {
        FT_LOG_WARNING("file %s only has %ld, but request %ld, loading model fails! \n",
                       filename.c_str(),
                       in_get_size,
                       loaded_data_size);
        return 0;
    }

    if (std::is_same<T, T_IN>::value == true) {
        cudaH2Dcpy(ptr, (T*)host_array.data(), size);
    }
    else {
        T_IN* ptr_2 = nullptr;
        deviceMalloc(&ptr_2, size, false);
        cudaH2Dcpy(ptr_2, host_array.data(), size);
        invokeCudaD2DcpyConvert(ptr, ptr_2, size);
        deviceFree(ptr_2);
    }
#ifdef AWS_SDK_ON
    if (filename.find("s3://") == 0) Aws::ShutdownAPI(options);
#endif
    return 0;
}

template int loadWeightFromBinFunc<float, float>(float* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<half, float>(half* ptr, std::vector<size_t> shape, std::string filename);
#ifdef ENABLE_BF16
template int
loadWeightFromBinFunc<__hip_bfloat16, float>(__hip_bfloat16* ptr, std::vector<size_t> shape, std::string filename);
#endif
template int loadWeightFromBinFunc<float, half>(float* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<half, half>(half* ptr, std::vector<size_t> shape, std::string filename);
#ifdef ENABLE_BF16
template int
loadWeightFromBinFunc<__hip_bfloat16, half>(__hip_bfloat16* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<float, __hip_bfloat16>(float* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<half, __hip_bfloat16>(half* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<__hip_bfloat16, __hip_bfloat16>(__hip_bfloat16*      ptr,
                                                                 std::vector<size_t> shape,
                                                                 std::string         filename);
#endif  // ENABLE_BF16

template<typename T>
int loadWeightFromBin(T* ptr, std::vector<size_t> shape, std::string filename, FtCudaDataType model_file_type)
{
    switch (model_file_type) {
        case FtCudaDataType::FP32:
            loadWeightFromBinFunc<T, float>(ptr, shape, filename);
            break;
        case FtCudaDataType::FP16:
            loadWeightFromBinFunc<T, half>(ptr, shape, filename);
            break;
#ifdef ENABLE_BF16
        case FtCudaDataType::BF16:
            loadWeightFromBinFunc<T, __hip_bfloat16>(ptr, shape, filename);
            break;
#endif
        default:
            FT_LOG_ERROR("Does not support FtCudaDataType=%d", model_file_type);
            FT_CHECK(false);
    }
    return 0;
}

template int
loadWeightFromBin(float* ptr, std::vector<size_t> shape, std::string filename, FtCudaDataType model_file_type);
template int
loadWeightFromBin(half* ptr, std::vector<size_t> shape, std::string filename, FtCudaDataType model_file_type);
#ifdef ENABLE_BF16
template int
loadWeightFromBin(__hip_bfloat16* ptr, std::vector<size_t> shape, std::string filename, FtCudaDataType model_file_type);
#endif

template<typename T_IN, typename T_OUT>
__global__ void cudaD2DcpyConvert(T_OUT* dst, const T_IN* src, const int size)
{
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x) {
        dst[tid] = convert_to_type<T_IN, T_OUT>(src[tid]);
    }
}

template<typename T_IN, typename T_OUT>
void invokeCudaD2DcpyConvert(T_OUT* tgt, const T_IN* src, const int size, hipStream_t stream)
{
    cudaD2DcpyConvert<<<256, 256, 0, stream>>>(tgt, src, size);
}

template void invokeCudaD2DcpyConvert(float* tgt, const float* src, const int size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(half* tgt, const float* src, const int size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, const half* src, const int size, hipStream_t stream);

#ifdef ENABLE_BF16
template void invokeCudaD2DcpyConvert(__hip_bfloat16* tgt, const float* src, const int size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, const __hip_bfloat16* src, const int size, hipStream_t stream);
#endif  // ENABLE_BF16

void invokeCudaD2DcpyHalf2Float(float* dst, half* src, const int size, hipStream_t stream)
{
    invokeCudaD2DcpyConvert(dst, src, size, stream);
}

void invokeCudaD2DcpyFloat2Half(half* dst, float* src, const int size, hipStream_t stream)
{
    invokeCudaD2DcpyConvert(dst, src, size, stream);
}

template<typename T>
void saveToBinary(const T* ptr, const int size, std::string filename)
{

    std::vector<T> h_ptr(size);
    cudaD2Hcpy(h_ptr.data(), ptr, size);
    std::vector<float> float_ptr(size);
    for (int i = 0; i < size; i++) {
        float_ptr[i] = (float)h_ptr[i];
    }

    std::ofstream out(filename, std::ios::out | std::ios::binary);
    FT_CHECK_WITH_INFO(out.is_open(), "Fail to open file " + filename);

    out.write((char*)float_ptr.data(), size * sizeof(float));
}

template void saveToBinary(const float* ptr, const int size, std::string filename);
template void saveToBinary(const half* ptr, const int size, std::string filename);
#ifdef ENABLE_BF16
template void saveToBinary(const __hip_bfloat16* ptr, const int size, std::string filename);
#endif  // ENABLE_BF16

template<>
void saveToBinary(const int* ptr, const int size, std::string filename)
{
    std::vector<int> h_ptr(size);
    cudaD2Hcpy(h_ptr.data(), ptr, size);
    std::ofstream out(filename, std::ios::out | std::ios::binary);
    FT_CHECK_WITH_INFO(out.is_open(), "Fail to open file " + filename);
    out.write((char*)h_ptr.data(), size * sizeof(int));
}

template<typename T_IN, typename T_fake_type>
__global__ void fakeCast(T_IN* input_ptr, const size_t size)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        T_fake_type tmp_val = (T_fake_type)((float)input_ptr[i]);
        tmp_val             = tmp_val * (T_fake_type)(1.0f);
        input_ptr[i]        = (T_IN)((float)tmp_val);
    }
}

template<typename T_IN, typename T_fake_type>
void invokeFakeCast(T_IN* input_ptr, const size_t size, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((size + 255) / 256);
    fakeCast<T_IN, T_fake_type><<<grid, block, 0, stream>>>(input_ptr, size);
}

#ifdef ENABLE_BF16
template void invokeFakeCast<float, __hip_bfloat16>(float* input_ptr, const size_t size, hipStream_t stream);
template void
invokeFakeCast<__hip_bfloat16, __hip_bfloat16>(__hip_bfloat16* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<half, __hip_bfloat16>(half* input_ptr, const size_t size, hipStream_t stream);
#endif
template void invokeFakeCast<float, half>(float* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<float, float>(float* input_ptr, const size_t size, hipStream_t stream);

}  // namespace fastertransformer
